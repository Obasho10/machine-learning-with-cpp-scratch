#include "hip/hip_runtime.h"
#include "layerconv.h"
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include<iostream>
#include <hip/hip_runtime.h>
#define pi 3.14159265359
hipError_t cudaStatus;


__global__ void conv_img_gpu(float *img, float *kernel, float *imgf, int kernel_size)
{
  int N=gridDim.x*blockDim.x;  
  //local ID of each thread (withing block) 
  int ix = threadIdx.x+blockIdx.x*blockDim.x;    
  //each block is assigned to a row of an image, iy index of y value                  
  int iy = threadIdx.y + blockIdx.y*blockDim.y;  
  //each thread is assigned to a pixel of a row, ix index of x value
  int iz = threadIdx.z ; 
  //idx global index (all blocks) of the image pixel 
  int idx = iz*N*N+iy*N +ix;                        
 //total number of kernel elements
  int K2 = kernel_size*kernel_size;  
  //center of kernel in both dimensions          
  int center = (kernel_size -1)/2;		 
  //Auxiliary variables
  int ii, jj;
  float sum = 0.0;
 /*
 Define a vector (float) sdata[] that will be hosted in shared memory,
 *extern* dynamic allocation of shared memory: kernel<<<blocks,threads,memory size to be allocated in shared memory>>>
*/  
/*
  Convlution of image with the kernel
  Each thread computes the resulting pixel value 
  from the convolution of the original image with the kernel;
  number of computations per thread = size_kernel^2
  The result is stored to imgf
  */
  if (idx<N*N*8)
  {
    for (int ki = 0; ki<kernel_size; ki++)
      for (int kj = 0; kj<kernel_size; kj++)
        for (int kk = 0;kk<8;kk++)
        {
            ii = (ix-center)+ki;
            jj = (iy-center)+kj;
            if(ii<0 || jj<0 || ii>N || jj>N)
            {
                sum+=0;
                continue;
            }
            sum+=img[kk*N*N+jj*N+ii]*kernel[kk*K2+kj*kernel_size + ki];
        }
    imgf[idx] = sum;
  }
}

layers::layers(int outSize, int inSize, int pad, int inLayers, int outLayers,int ker) 
    : outputSize(outSize), inputSize(inSize), padding(pad), inputLayers(inLayers), outputLayers(outLayers),kernal(ker) {
    for (int i = 0; i < 2; ++i) {
        hipMalloc(&inputMatricesGPU[i], inputLayers * inputSize * inputSize * sizeof(float));
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) 
        {
            std::cout<<hipGetErrorString(cudaStatus);
        }
        else
            std::cout<<"sucess copy"<<std::endl;
        hipMalloc(&outputMatricesGPU[i], outputLayers * outputSize * outputSize * sizeof(float));
    }
    hipMalloc(&inputWeightMatricesGPU, inputLayers * inputLayers * kernal*kernal * sizeof(float));
    hipMalloc(&outputWeightMatricesGPU, outputLayers * inputLayers * kernal*kernal * sizeof(float));
    cudaStatus = hipGetLastError();
    std::cout<<'0';
    if (cudaStatus != hipSuccess) 
    {
        std::cout<<hipGetErrorString(cudaStatus);
    }
    else
        std::cout<<"sucess copy"<<std::endl;
    calculate_weights(kernal,inputLayers, 1,inputWeightMatricesGPU);
    std::cout<<'0';
    calculate_weights(kernal,outputLayers,1, outputWeightMatricesGPU);
    

}

layers::~layers() {
    for (int i = 0; i < 2; ++i) {
        hipFree(inputMatricesGPU[i]);
        hipFree(outputMatricesGPU[i]);
    }
    hipFree(outputWeightMatricesGPU);
    hipFree(inputWeightMatricesGPU);
}

void layers::copyInputToGPU(const std::vector<std::vector<std::vector<float>>>& inputData, int index) {
    hipMemcpy(inputMatricesGPU[index], inputData.data(), inputLayers * inputSize * inputSize * sizeof(float), hipMemcpyHostToDevice);
}

void layers::copyOutputToGPU(const std::vector<std::vector<std::vector<float>>>& outputData, int index) {
    hipMemcpy(outputMatricesGPU[index], outputData.data(), outputLayers * outputSize * outputSize * sizeof(float), hipMemcpyHostToDevice);
}

void layers::copyInputToCPU(std::vector<std::vector<std::vector<float>>>& inputData, int index) {
    hipMemcpy(inputData.data(), inputMatricesGPU[index], inputLayers * inputSize * inputSize * sizeof(float), hipMemcpyDeviceToHost);
}

void layers::copyOutputToCPU(std::vector<std::vector<std::vector<float>>>& outputData, int index) {
    hipMemcpy(outputData.data(), outputMatricesGPU[index], outputLayers * outputSize * outputSize * sizeof(float), hipMemcpyDeviceToHost);
}

void layers::copyInputWeightsToGPU(const std::vector<std::vector<std::vector<float>>>& inputWeights) {
    hipMemcpy(inputWeightMatricesGPU, inputWeights.data(), inputLayers * inputLayers * kernal*kernal * sizeof(float), hipMemcpyHostToDevice);
}

void layers::copyOutputWeightsToGPU(const std::vector<std::vector<std::vector<float>>>& outputWeights) {
    hipMemcpy(outputWeightMatricesGPU, outputWeights.data(), outputLayers * outputSize * outputSize * sizeof(float), hipMemcpyHostToDevice);
}

void layers::copyInputWeightsToCPU(std::vector<std::vector<std::vector<float>>>& inputWeights) {
    hipMemcpy(inputWeights.data(), inputMatricesGPU,inputLayers * inputLayers * kernal*kernal * sizeof(float), hipMemcpyDeviceToHost);
}

void layers::copyOutputWeightsToCPU(std::vector<std::vector<std::vector<float>>>& outputWeights) {
    hipMemcpy(outputWeights.data(), outputMatricesGPU, outputLayers * outputSize * outputSize * sizeof(float), hipMemcpyDeviceToHost);
}

void layers::InputConv(int padding)
{
    dim3 blockSize;
    if(inputSize<8)
        dim3 blockSize(inputSize,inputSize,8);
    else
        dim3 blockSize(8,8,8);
    std::cout<<blockSize.x;
    dim3 gridSize(inputSize/blockSize.x,inputSize/blockSize.y,1);
    conv_img_gpu<<<gridSize, blockSize>>>(inputMatricesGPU[0],inputWeightMatricesGPU, inputMatricesGPU[1], kernal);
    hipDeviceSynchronize(); // Wait for kernel to finish
}

void layers::calculate_weights(int kernel_size, int inlayers, float sigma, float *kernel)
{
    int Nk3 = kernel_size * kernel_size * inputLayers;
    float center = (kernel_size - 1) / 2.0f;
    for (int i = 0; i < Nk3; i++)
    {
        int z_idx = i / (kernel_size * kernel_size);
        int rem = i % (kernel_size * kernel_size);
        int y_idx = rem / kernel_size;
        int x_idx = rem % kernel_size;

        float x = x_idx - center;
        float y = y_idx - center;
        float z = z_idx - center;

        kernel[i] = -(1.0f / (pi * pow(sigma, 6))) * (1.0f - 0.5f * ((x * x + y * y + z * z) / (sigma * sigma))) * exp(-0.5f * ((x * x + y * y + z * z) / (sigma * sigma)));
    }
}
